/// Trabajo Práctico 3. Programación paralela de la multiplicación de matrices

/// Implementación en un coprocesador de tipo GPU usando CUDA

// Inclusiones
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



// Definiciones
#define N 2
#define BLOCK_SIZE 2

// Funciones
__global__ void multiplicar_matrices(int *a, int *b, int *c, int n) {
	int fila = blockIdx.y * blockDim.y + threadIdx.y;
	int columna = blockIdx.x * blockDim.x + threadIdx.x;

	if (fila < n && columna < n) {
		int suma = 0;
		for (int i = 0; i < n; i++) {
			suma += a[fila * n + i] * b[i * n + columna];
		}
		c[fila * n + columna] = suma;
	}
}


// Main
int main() {
	int *a_cpu, *b_cpu, *c_cpu;
	int *a_gpu, *b_gpu, *c_gpu;
	size_t size = N * N * sizeof(int);

	// Reserva de memoria en CPU
	a_cpu = (int *)malloc(size);
	b_cpu = (int *)malloc(size);
	c_cpu = (int *)malloc(size);

	// Inicializar matrices
	for (int i = 0; i < N * N; i++) {
		a_cpu[i] = rand() % 10;
		b_cpu[i] = rand() % 10;
	}

	// Mostrar matrices
	printf("Matriz A:\n");
	for (int i = 0; i < N * N; i++) {
		printf("%d ", a_cpu[i]);
		if ((i + 1) % N == 0) {
			printf("\n");
		}
	}
	printf("\n");

	printf("Matriz B:\n");
	for (int i = 0; i < N * N; i++) {
		printf("%d ", b_cpu[i]);
		if ((i + 1) % N == 0) {
			printf("\n");
		}
	}
	printf("\n");

	// Reserva de memoria en GPU
	hipMalloc(&a_gpu, size);
	hipMalloc(&b_gpu, size);
	hipMalloc(&c_gpu, size);

	// Copiar datos de CPU a GPU
	hipMemcpy(a_gpu, a_cpu, size, hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b_cpu, size, hipMemcpyHostToDevice);

	// Definir bloques e hilos
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

	// Lanzar kernel
	multiplicar_matrices<<<numBlocks, threadsPerBlock>>>(a_gpu, b_gpu, c_gpu, N);

	// Copiar datos de GPU a CPU
	hipMemcpy(c_cpu, c_gpu, size, hipMemcpyDeviceToHost);

	// Mostrar resultado
	printf("Matriz C:\n");
	for (int i = 0; i < N * N; i++) {
		printf("%d ", c_cpu[i]);
		if ((i + 1) % N == 0) {
			printf("\n");
		}
	}
	printf("\n");

	// Liberar memoria
	free(a_cpu);
	free(b_cpu);
	free(c_cpu);
	hipFree(a_gpu);
	hipFree(b_gpu);
	hipFree(c_gpu);

	return 0;
}